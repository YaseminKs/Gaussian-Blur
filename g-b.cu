#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>

#define BLOCK_SIZE 16
__constant__ float d_kernel[9] = { 1/16.0f, 2/16.0f, 1/16.0f, 2/16.0f, 4/16.0f, 2/16.0f, 1/16.0f, 2/16.0f, 1/16.0f };

__global__ void gaussianBlurKernel( unsigned char *input, unsigned char *output, int width, int height, int channels ){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if( x >= 1 && y >= 1 && x < width - 1 && y < height - 1 ){
        for( int c = 0 ; c < channels ; c++ ){
            float sum = 0.0f;
            for( int i = -1 ; i <= 1 ; i++ ){
                for( int j = -1 ; j <= 1 ; j++ ){
                    int idx = ( ( y + i ) * width + ( x + j ) ) * channels + c;
                    sum += input[idx] * d_kernel[( i + 1 ) * 3 + ( j + 1 )];
                }
            }
            output[( y * width + x ) * channels + c] = ( unsigned char )sum;
        }
    }
}

void applyGaussianBlurCUDA( cv::Mat &image, cv::Mat &output ){
    int imgSize = image.rows * image.cols * image.channels();
    unsigned char *d_input, *d_output;

    hipMalloc( ( void** )&d_input, imgSize );
    hipMalloc( ( void** )&d_output, imgSize );
    hipMemcpy( d_input, image.data, imgSize, hipMemcpyHostToDevice );

    dim3 blockSize( BLOCK_SIZE, BLOCK_SIZE );
    dim3 gridSize( ( image.cols + BLOCK_SIZE - 1 ) / BLOCK_SIZE, ( image.rows + BLOCK_SIZE - 1 ) / BLOCK_SIZE );
    
    gaussianBlurKernel<<<gridSize, blockSize>>>( d_input, d_output, image.cols, image.rows, image.channels() );

    hipMemcpy( output.data, d_output, imgSize, hipMemcpyDeviceToHost );

    hipFree( d_input );
    hipFree( d_output );
}

int main(){
    cv::Mat image = cv::imread( "input.jpg" );
    if( image.empty() ){
        std::cout << "Error loading image!" << std::endl;
        return -1;
    }

    cv::Mat output( image.size(), image.type() );
    applyGaussianBlurCUDA( image, output );
    cv::imwrite( "output.jpg", output );

    std::cout << "Gaussian blur applied using CUDA!" << std::endl;
    return 0;
}
